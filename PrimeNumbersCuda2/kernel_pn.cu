﻿
#include "hip/hip_runtime.h"

#include <list>
#include <stdio.h>
#include <string>
#include <iostream>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, const int h, int *p);

__global__ void addKernel(int *c, const int *a, const int *b, const int *h, int *p)
// musimy miec low, high i 
{
    printf("Hello, prime number will be generated from the gpu device!\n");

    int low = 0, ii;
    const int high = *h;
    bool isPrime = true;
    int pi = 0;
    int pp[20] = { 0 };




    for (int low = 0; low < high; low++) {

        if (low <= 1) {
            isPrime = false;
            continue;
        }
        else if (low == 3) {
            isPrime = true;
            p[pi] = low;
            pi = pi + 1;
            continue;
        }
        else if (low % 2 == 0 | low % 3 == 0) {
            isPrime = false;
            continue;
        }

        int iy = 5;

        for (int ii = iy; ii * ii <= low; ii++) {

            if (low % ii == 0 | low % (ii + 2) == 0) {
                isPrime = false;
                continue;
            }
            ii = ii + 6;
        }
        isPrime = true;
        p[pi] = low;
        pi = pi + 1;

    }




    //while (low < high) {
    //    isPrime = true;
    //    if (low == 0 || low == 1) {
    //        isPrime = false;
    //    }
    //    else {
    //        for (ii = 2; ii <= low / 2; ++ii) {
    //            if (low % ii == 0) {
    //                isPrime = false;
    //                break;
    //            }
    //        }
    //    }

    //    if (isPrime) {
    //        printf("%d\n", low);
    //        p[pi] = low;
    //       pi = pi + 1;
    //       //printf("print ", pi);
 
    //    }

    //    ++low;
    //}






    //p = pp;
    printf("kuk");
    int loop;
    for (loop = 0; loop < 20; loop++)
        printf("%d ", p[loop]);
    printf("kuk");

}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    const int high = 15;
    int p_main[20] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, high, p_main);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // drukowanie listy prime
    printf("fffffffffffffffff");
    int loop;
    for (loop = 0; loop < 20; loop++)
        printf("%d ", p_main[loop]);
    printf("kuk");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, const int h, int *p)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    int *hh = 0;
    int* pp = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&pp, 20 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&hh, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(hh, &h, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 1>>>(dev_c, dev_a, dev_b, hh, pp);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
   cudaStatus = hipMemcpy(p, pp, 20 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
